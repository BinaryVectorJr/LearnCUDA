#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define N 10000000
#define MAX_ERR 1e-6

/// CUDA Hello World
//__global__ void cuda_hello()
//{
//	printf("Hello World from GPU");
//}

/// CPU Add Vectors
//void vector_add(float* out, float* a, float* b, int n)
//{
//	for (int i = 0; i < n; i++)
//	{
//		out[i] = a[i] + b[i];
//	}
//}

// CUDA Vector Addition
__global__ void vector_add_cuda(float* out, float* a, float* b, int n)
{
	for (int i = 0; i < n; i++)
	{
		out[i] = a[i] + b[i];
	}
}

int main()
{
	/// CUDA Hello World
		//cuda_hello <<<1, 1 >>> ();
		//return 0;
	///-----------------------------------
	
	/// CPU Vector Add
		//float* a, * b, * out;

		//// Allocate memory
		//a = (float*)malloc(sizeof(float) * N);
		//b = (float*)malloc(sizeof(float) * N);
		//out = (float*)malloc(sizeof(float) * N);

		////Initialize Array
		//for (int i = 0; i < N; i++)
		//{
		//	a[i] = 1.0f;
		//	b[i] = 2.0f;
		//}

		//// Call Vector Add main function
		//vector_add(out, a, b, N);
	///-------------------------------------

	/// CUDA Vector Add
	float* a, * b, * out;
	float* dev_a, *dev_b, *dev_out;

	// Allotting the host memory for array a, b and the output array
	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	/// MISSED - Initializing Host Arrays
	for (int i = 0; i < N; i++)
	{
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	// Allotting the device memory for array a, b, and output array
	hipMalloc((void**)&dev_a, sizeof(float) * N);
	hipMalloc((void**)&dev_b, sizeof(float) * N);
	hipMalloc((void**)&dev_out, sizeof(float) * N);

	// Transfer data from host memory to device memory
	hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	vector_add_cuda << <1, 1 >> > (dev_out, dev_a, dev_b, N);

	// Transfer data back from device memory to host memory
	hipMemcpy(out, dev_out, sizeof(float) * N, hipMemcpyDeviceToHost);

	// MISSED - Verification
	for (int i = 0; i < N; i++)
	{
		assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
	}
	printf("out[0] = %f\n", out[0]);
	printf("PASSED\n");

	// Cleaning up the pointers (deallocating) after kernel execution
	// Need to do for both host and device
	
	// Deallocate device memory first (together) then host memory (together)
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_out);

	// Deallocate host memory
	free(a);
	free(b);
	free(out); // MISSED
}